#include "hip/hip_runtime.h"
#include "similarity_check.hpp"
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>


using namespace std;
using namespace cv;
using namespace simcheck;


__host__ void CHECK_LAST_CUDA_ERROR(const char * kernel_name)
{
    hipError_t cudaerr {hipGetLastError()};

    printf("----- Kernel \"%s\" ----- \n", kernel_name);
    if (cudaerr != hipSuccess){
        printf("    CUDA Runtime Error at \"%s\".\n", hipGetErrorString(cudaerr));
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
    else
        printf("    No CUDA Runtime Error (No Synchronous Error)\n\n\n");

}

__host__ void CHECK_CUDA_ASYNC_ERROR(const char * kernel_name)
{
    hipError_t cudaerr {hipDeviceSynchronize()};
    //printf("Kernel \"%s\": \n", kernel_name);
    printf("----- Kernel \"%s\" ----- \n", kernel_name);
    if (cudaerr != hipSuccess){
        printf("    kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    }
    else
        printf("    Successfully launch the kernel (No Asynchronous Error)\n\n\n");

}

__host__ void similarity_check::execute(const char * image_1_path,
                                        const char * image_2_path)
{   
    Mat img_1 = imread(image_1_path, IMREAD_COLOR);
    Mat img_2 = imread(image_2_path, IMREAD_COLOR);

    int img_1_w = img_1.size().width;
    int img_1_h = img_1.size().height;

    int img_2_w = img_2.size().width;
    int img_2_h = img_2.size().height;

    printf("Image: %s\n", image_1_path);
    printf("    width: %d\n", img_1_w);
    printf("    height: %d\n", img_1_h);
    printf("    type: %d\n", img_1.type());
 
    printf(" debug: %d\n", *img_1.data);

    printf("Image: %s\n", image_2_path);
    printf("    width: %d\n", img_2_w);
    printf("    height: %d\n", img_2_h);
    printf("    type: %d\n", img_2.type());


    if(img_1.empty())
    {
        cout << "Could not read the image: " << image_1_path << endl;
        return;
    }

    if(img_2.empty())
    {
        cout << "Could not read the image: " << image_2_path << endl;
        return;
    }


    if(img_1_w != img_2_w || img_1_h != img_2_h)
    {
        cout << "Two images are not in the same size." << endl;
    }


    img_height = img_1_h;
    img_width  = img_1_w;


    //imshow("Display window", img_2);


    //int k = waitKey(0); // Wait for a keystroke in the window
    //if(k == 's')
    //{
    //    imwrite("starry_night.png", img_1);
    //}


    unsigned long long N = img_height * img_width * channel;
    uint8_t *d_img_1, *d_img_2, *d_diff;

    hipMalloc((void**)&d_img_1, sizeof(uint8_t) * N);
    hipMalloc((void**)&d_img_2, sizeof(uint8_t) * N);
    hipMalloc((void**)&d_diff,  sizeof(uint8_t) * N);
    uint8_t * res_diff = (uint8_t *)malloc(sizeof(uint8_t)*N);

    hipMemcpy(d_img_1, img_1.data, sizeof(uint8_t) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_img_2, img_2.data, sizeof(uint8_t) * N, hipMemcpyHostToDevice);
    cout << "Transfer image 1 and image 2 to GPU Memory" << endl;


    //const dim3 numBlocks(99, 45);
    //const dim3 threadsPerBlock(18, 18);a

    constexpr int block_size {32};
    printf("DIM: %ld %ld\n",(block_size+img_width-1)/block_size, (block_size+img_height-1)/block_size);
    const dim3 numBlocks((block_size+img_width-1)/block_size, (block_size+img_height-1)/block_size);
    const dim3 threadsPerBlock(block_size, block_size);


    compute_psnr_cuda<<<numBlocks,threadsPerBlock>>>(d_diff,
                                                     d_img_1,
                                                     d_img_2,
                                                     img_1_h,
                                                     img_1_w);

    CHECK_LAST_CUDA_ERROR("PSRN Calculation");
    CHECK_CUDA_ASYNC_ERROR("Calculation");


    hipMemcpy(res_diff, d_diff, sizeof(uint8_t) * N, hipMemcpyDeviceToHost);


    //printf("test res: %d\n", res_diff[0]);

    //uint8_t * tmp_img_1 = img_1.data;
    //uint8_t * tmp_img_2 = img_2.data;
    //for(int i=0;i<img_1_h;++i){
    //    for(int j=0;j<3*img_1_w;j+=3){
    //        if(res_diff[i*3*img_1_w+j+0] !=0 || res_diff[i*3*img_1_w+j+1] !=0 || res_diff[i*3*img_1_w+j+2] !=0){
    //            printf("Pixel[%d][%d]:\n",i,j);
    //            printf("    R: %d\n",res_diff[i*3*img_1_w+j+0]);
    //            printf("    G: %d\n",res_diff[i*3*img_1_w+j+1]);
    //            printf("    B: %d\n",res_diff[i*3*img_1_w+j+2]);
    //            printf("CPU R: %d\n",abs(tmp_img_1[i*3*img_1_w+j+0] - tmp_img_2[i*3*img_1_w+j+0]));
    //            printf("CPU G: %d\n",abs(tmp_img_1[i*3*img_1_w+j+1] - tmp_img_2[i*3*img_1_w+j+1]));
    //            printf("CPU B: %d\n",abs(tmp_img_1[i*3*img_1_w+j+2] - tmp_img_2[i*3*img_1_w+j+2]));

    //        }
    //    }
    //}

    //uint8_t * tmp_img_1 = img_1.data;
    //uint8_t * tmp_img_2 = img_2.data;
    //for(int i=0;i<img_1_h;++i){
    //    for(int j=0;j<3*img_1_w;j+=3){
    //        if(tmp_img_1[i*3*img_1_w+j+0] != tmp_img_2[i*3*img_1_w+j+0]){
    //            printf("Pixel[%d][%d] is different\n", i,j/3);
    //            printf("    img_1: %d\n",tmp_img_1[i*3*img_1_w+j+0]);
    //            printf("    img_2: %d\n",tmp_img_2[i*3*img_1_w+j+0]);
    //            printf("    diff: %d\n",res_diff[i*3*img_1_w+j+0]);

    //        }
    //    }
    //}


    hipFree(d_img_1);
    hipFree(d_img_2);
    hipFree(d_diff);
    return;
}

__global__ void compute_psnr_cuda(uint8_t *d_diff,
                                  uint8_t *d_img_1,
                                  uint8_t *d_img_2,
                                  unsigned int height,
                                  unsigned int width)
{

    const int x = threadIdx.x+blockIdx.x*blockDim.x;
    const int y = threadIdx.y+blockIdx.y*blockDim.y;

    if(x>=width || y>=height) return;

    d_diff[x+y*width*3+0] = abs(d_img_1[x+y*width*3+0]-d_img_2[x+y*width*3+0]);
    d_diff[x+y*width*3+1] = abs(d_img_1[x+y*width*3+1]-d_img_2[x+y*width*3+1]);
    d_diff[x+y*width*3+2] = abs(d_img_1[x+y*width*3+2]-d_img_2[x+y*width*3+2]);
}
